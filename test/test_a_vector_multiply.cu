#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include "vector.h"
#include "matrix.h"

int main(int argc, char **argv) {
	float *ys = (float *) malloc(64 * 64 * sizeof(float));

	for (int i = 0; i < 64 * 64; ++i)
		ys[i] = 2.0;

	dt::matrix<64, 64> A;
	dt::vector<64 * 64> x(ys), y = A * x;
	CHECK(hipGetLastError());

	y.get(ys);

	for (int i = 0; i < 64 * 64; ++i)
		printf("%f\n", ys[i]);

	free(ys);
}
