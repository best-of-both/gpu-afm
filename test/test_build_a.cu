#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include "build_a.h"

#define BLOCK_SIZE 1024 //@@ Number of threads per block
#define DEBUG 0         //@@ toggle for debug messages
#define REPS 1		//Max number of runs for timing analysis

/* Choose which kernel to run:
  1) Naive implementation
  2) improved branch performance (non-divergent)
  3) Non Divergent  total w/ GPU summing
  4) Sequential Addressing
*/
#define KERNEL_SELECT 1

char *inputFile;
char *outputFile;

#define CHECK(e) _errorCheck(e, __FILE__, __LINE__)

void _errorCheck(hipError_t e, const char* file, int line){
	if(e != hipSuccess){
		printf("Failed to run statement: %s %d \n", file, line);
		exit(1);
	}
}


int main(int argc, char **argv) {

	int* h_a;
	int* d_a;

	const unsigned int nx = 64;
	const unsigned int ny = 64;

	CHECK(hipMalloc((void**) &d_a, 4 * nx * ny * sizeof(int)));
	h_a = (int*) malloc(4 * nx * ny * sizeof(int));

	unsigned int grid_x = nx * ny / 1024;

	device_build_a<<<grid_x, 1024>>>(d_a, nx, ny);
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(h_a, d_a, 4 * nx * ny * sizeof(int), hipMemcpyDeviceToHost));

	for (unsigned int i = 0; i < nx * ny; ++i)
		printf("%d %d -4 %d %d\n", h_a[4 * i + 0], h_a[4 * i + 1], h_a[4 * i + 2], h_a[4 * i + 3]);
}
