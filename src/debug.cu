#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "debug.cuh"

void _cudaErrorCheck(hipError_t e, const char* file, int line){
	if(e != hipSuccess){
		printf("Failed to run statement (%s:%d): %s \n",
		       file, line, hipGetErrorString(e));
		exit(1);
	}
}
