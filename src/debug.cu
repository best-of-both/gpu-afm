
#include <hip/hip_runtime.h>
#define CHECK(e) _errorCheck(e, __FILE__, __LINE__)

void _errorCheck(hipError_t e, const char* file, int line){
	if(e != hipSuccess){
		printf("Failed to run statement (%s:%d): %s \n",
		       file, line, hipGetErrorString(e));
		exit(1);
	}
}
